#include "..\common\book.h"
#include "hip/hip_runtime.h"
#include ""

#define  SIZE (100*1024*1024)

float cpumalloc(int size,bool up)
{
  hipEvent_t start,stop;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  int *a,*dev_a;
  a=(int *)malloc(SIZE*sizeof(*a));
  HANDLE_NULL(a);
  HANDLE_ERROR(hipMalloc((void **)&dev_a,size*sizeof(*dev_a)));
  HANDLE_ERROR(hipEventRecord(start,0));
  for (int i=0;i!=100;++i)
  {
    if (up)
    {
      HANDLE_ERROR(hipMemcpy(a,dev_a,size*sizeof(*a),hipMemcpyDeviceToHost));
    } 
    else
    {
      HANDLE_ERROR(hipMemcpy(dev_a,a,size*sizeof(*a),hipMemcpyHostToDevice));
    }
  }
  HANDLE_ERROR(hipEventRecord(stop,0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  float elaspedtime;
  HANDLE_ERROR(hipEventElapsedTime(&elaspedtime,start,stop));
  HANDLE_ERROR(hipFree(dev_a));
  free(a);
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));
  return elaspedtime;
}

float cudahostmalloc(int size,bool up)
{
  hipEvent_t start,stop;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  int *a,*dev_a;
  HANDLE_ERROR(hipMalloc((void **)&dev_a,size*sizeof(*dev_a)));
  HANDLE_ERROR(hipHostAlloc((void **)&a,size*sizeof(*a),hipHostMallocDefault));
  HANDLE_ERROR(hipEventRecord(start,0));
  for (int i=0;i!=100;++i)
  {
    if (up)
    {
      HANDLE_ERROR(hipMemcpy(a,dev_a,size*sizeof(*a),hipMemcpyDeviceToHost));
    } 
    else
    {
      HANDLE_ERROR(hipMemcpy(dev_a,a,size*sizeof(*a),hipMemcpyHostToDevice));
    }
  }
  HANDLE_ERROR(hipEventRecord(stop,0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  float elaspedtime;
  HANDLE_ERROR(hipEventElapsedTime(&elaspedtime,start,stop));
  HANDLE_ERROR(hipHostFree(a));
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));
  return elaspedtime;
}

int main(void)
{
  float elaspedtime;
  float MB=(float)100*sizeof(int)*SIZE/1024./1024.;
  bool up=true;
  elaspedtime=cpumalloc(SIZE,up);
  printf("device to host: \nrun time using malloc:%f ms\n",elaspedtime);
  printf("%f MB per second.\n",MB/elaspedtime/1000.);
  elaspedtime=cudahostmalloc(SIZE,up);
  printf("run time using hostmalloc:%f ms\n",elaspedtime);
  printf("%f MB per second.\n",MB/elaspedtime/1000.);
  up=false;
  elaspedtime=cpumalloc(SIZE,up);
  printf("\n host to device: \nrun time using malloc:%f ms\n",elaspedtime);
  printf("%f MB per second.\n",MB/elaspedtime/1000.);
  elaspedtime=cudahostmalloc(SIZE,up);
  printf("run time using hostmalloc:%f ms\n",elaspedtime);
  printf("%f MB per second.\n",MB/elaspedtime/1000.);
  getchar();
  return 0;
}