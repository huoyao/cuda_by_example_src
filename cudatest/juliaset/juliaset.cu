#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "book.h"
#include "cpu_bitmap.h"
#include <hip/hip_runtime.h>
#include""

#define DIM 1000
struct hipComplex 
{
  float r;
  float i;
  __device__ hipComplex(float a,float b):r(a),i(b){}
  __device__ float magnitude2(void){return r*r+i*i;}
  __device__ hipComplex operator*(const hipComplex &a){return hipComplex(r*a.r-i*a.i,i*a.r+r*a.i);}
  __device__ hipComplex operator+(const hipComplex &a){return hipComplex(r+a.r,i+a.i);}
};

__device__ int julia(int x,int y)
{
  const float scale_=1.5;
  float jx=scale_*(float)(DIM/2-x)/(DIM/2);
  float jy=scale_*(float)(DIM/2-y)/(DIM/2);
  hipComplex c(-0.8,0.156);
  hipComplex a(jx,jy);
  for (int i=0;i!=200;++i)
  {
    a=a*a+c;
    if (a.magnitude2()>1000)
    {
      return 0;
    }
  }
  return 1;
}

__global__ void kernel(unsigned char *ptr)
{
  int x=blockIdx.x;
  int y=blockIdx.y;
  int offSet=x+y*gridDim.x;
  int juliavale=julia(x,y);
  ptr[offSet*4+0]=255*juliavale;
  ptr[offSet*4+1]=0;
  ptr[offSet*4+2]=0;
  ptr[offSet*4+3]=255;
}

struct DataBlock {
  unsigned char *device_pt;
};

int main(void)
{
  //DataBlock data;
  //CPUBitmap bitmap(DIM,DIM,&data);
  CPUBitmap bitmap(DIM,DIM);
  unsigned char *dev_bitmap;
  HANDLE_ERROR(hipMalloc((void**)&dev_bitmap,bitmap.image_size()));
  //data.device_pt=dev_bitmap;
  dim3 grid(DIM,DIM);
  kernel<<<grid,1>>>(dev_bitmap);
  HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(),dev_bitmap,bitmap.image_size(),hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipFree(dev_bitmap));
  bitmap.display_and_exit();
  hipDeviceReset();
}