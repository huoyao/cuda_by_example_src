#include "hip/hip_runtime.h"
#include "..\common\book.h"
#include "hip/hip_runtime.h"
#include ""

#define  N 1024*1024
#define FULL_SIZE (N*20)

__global__ void kernel(int *a,int *b,int *c)
{
  int i=threadIdx.x+blockIdx.x*blockDim.x;
  if (i<N)
  {
    int id1=(i+1)%256;
    int id2=(i-1)%256;
    a[i]=(a[id1]+a[id2]+a[i])/3.0f;
    b[i]=(b[id1]+b[id2]+b[i])/3.0f;
    c[i]=(a[i]+b[i])/2.0f;
  }
}

int main(void)
{
  hipDeviceProp_t prop;
  int dev;
  HANDLE_ERROR(hipGetDevice(&dev));
  HANDLE_ERROR(hipGetDeviceProperties(&prop,dev));
  if (!prop.deviceOverlap)
  {
    printf("device can`t overflap\n");
    return 0;
  }
  hipEvent_t start,stop;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  HANDLE_ERROR(hipEventRecord(start,0));
  hipStream_t stream;
  HANDLE_ERROR(hipStreamCreate(&stream));
  int *dev_a,*dev_b,*dev_c;
  int *a,*b,*c;
  HANDLE_ERROR(hipMalloc((void **)&dev_a,N*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_b,N*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_c,N*sizeof(int)));
  HANDLE_ERROR(hipHostAlloc((void **)&a,FULL_SIZE*sizeof(int),hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void **)&b,FULL_SIZE*sizeof(int),hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void **)&c,FULL_SIZE*sizeof(int),hipHostMallocDefault));
  for (int i=0;i!=FULL_SIZE;++i)
  {
    a[i]=rand();
    b[i]=rand();
  }
  for (int i=0;i!=FULL_SIZE;i+=N)
  {
    HANDLE_ERROR(hipMemcpyAsync(dev_a,a+i,N*sizeof(int),hipMemcpyHostToDevice,stream));
    HANDLE_ERROR(hipMemcpyAsync(dev_b,b+i,N*sizeof(int),hipMemcpyHostToDevice,stream));
    kernel<<<N/256,256,0,stream>>>(dev_a,dev_b,dev_c);
    HANDLE_ERROR(hipMemcpyAsync(c+i,dev_c,N*sizeof(int),hipMemcpyDeviceToHost,stream));
  }
  HANDLE_ERROR(hipStreamSynchronize(stream));
  float elaspedtime;
  HANDLE_ERROR(hipEventRecord(stop,0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elaspedtime,start,stop));
  printf("total time:%.3f ms\n",elaspedtime);
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));
  HANDLE_ERROR(hipHostFree(a));
  HANDLE_ERROR(hipHostFree(b));
  HANDLE_ERROR(hipHostFree(c));
  HANDLE_ERROR(hipStreamDestroy(stream));
  getchar();
  return 0;
}