#include "hip/hip_runtime.h"
#include "..\common\book.h"
#include "hip/hip_runtime.h"
#include ""

#define  N 1024*1024
#define FULL_SIZE (N*20)

__global__ void kernel(int *a,int *b,int *c)
{
  int i=threadIdx.x+blockIdx.x*blockDim.x;
  if (i<N)
  {
    int id1=(i+1)%256;
    int id2=(i-1)%256;
    a[i]=(a[id1]+a[id2]+a[i])/3.0f;
    b[i]=(b[id1]+b[id2]+b[i])/3.0f;
    c[i]=(a[i]+b[i])/2.0f;
  }
}

int main(void)
{
  hipDeviceProp_t prop;
  int dev;
  HANDLE_ERROR(hipGetDevice(&dev));
  HANDLE_ERROR(hipGetDeviceProperties(&prop,dev));
  if (!prop.deviceOverlap)
  {
    printf("device can`t overflap\n");
    return 0;
  }
  hipEvent_t start,stop;
  HANDLE_ERROR(hipEventCreate(&start));
  HANDLE_ERROR(hipEventCreate(&stop));
  hipStream_t stream,stream1;
  HANDLE_ERROR(hipStreamCreate(&stream));
  HANDLE_ERROR(hipStreamCreate(&stream1));
  int *dev_a,*dev_b,*dev_c;
  int *dev_a1,*dev_b1,*dev_c1;
  int *a,*b,*c;
  HANDLE_ERROR(hipMalloc((void **)&dev_a,N*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_b,N*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_c,N*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_a1,N*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_b1,N*sizeof(int)));
  HANDLE_ERROR(hipMalloc((void **)&dev_c1,N*sizeof(int)));
  HANDLE_ERROR(hipHostAlloc((void **)&a,FULL_SIZE*sizeof(int),hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void **)&b,FULL_SIZE*sizeof(int),hipHostMallocDefault));
  HANDLE_ERROR(hipHostAlloc((void **)&c,FULL_SIZE*sizeof(int),hipHostMallocDefault));
  for (int i=0;i!=FULL_SIZE;++i)
  {
    a[i]=rand();
    b[i]=rand();
  }
  //first time
  HANDLE_ERROR(hipEventRecord(start,0));
  for (int i=0;i!=FULL_SIZE;i+=2*N)
  {
    HANDLE_ERROR(hipMemcpyAsync(dev_a,a+i,N*sizeof(int),hipMemcpyHostToDevice,stream));
    HANDLE_ERROR(hipMemcpyAsync(dev_b,b+i,N*sizeof(int),hipMemcpyHostToDevice,stream));
    kernel<<<N/256,256,0,stream>>>(dev_a,dev_b,dev_c);
    HANDLE_ERROR(hipMemcpyAsync(c+i,dev_c,N*sizeof(int),hipMemcpyDeviceToHost,stream));

    HANDLE_ERROR(hipMemcpyAsync(dev_a1,a+i+N,N*sizeof(int),hipMemcpyHostToDevice,stream1));
    HANDLE_ERROR(hipMemcpyAsync(dev_b1,b+i+N,N*sizeof(int),hipMemcpyHostToDevice,stream1));
    kernel<<<N/256,256,0,stream1>>>(dev_a1,dev_b1,dev_c1);
    HANDLE_ERROR(hipMemcpyAsync(c+i+N,dev_c1,N*sizeof(int),hipMemcpyDeviceToHost,stream1));
  }
  float elaspedtime;
  HANDLE_ERROR(hipEventRecord(stop,0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elaspedtime,start,stop));
  printf("first total time:%.3f ms\n",elaspedtime);
  //second time
  HANDLE_ERROR(hipEventRecord(start,0));
  for (int i=0;i!=FULL_SIZE;i+=2*N)
  {
    HANDLE_ERROR(hipMemcpyAsync(dev_a,a+i,N*sizeof(int),hipMemcpyHostToDevice,stream));
    HANDLE_ERROR(hipMemcpyAsync(dev_a1,a+i+N,N*sizeof(int),hipMemcpyHostToDevice,stream1));
    HANDLE_ERROR(hipMemcpyAsync(dev_b,b+i,N*sizeof(int),hipMemcpyHostToDevice,stream));
    HANDLE_ERROR(hipMemcpyAsync(dev_b1,b+i+N,N*sizeof(int),hipMemcpyHostToDevice,stream1));
    kernel<<<N/256,256,0,stream>>>(dev_a,dev_b,dev_c);
    kernel<<<N/256,256,0,stream1>>>(dev_a1,dev_b1,dev_c1);
    HANDLE_ERROR(hipMemcpyAsync(c+i,dev_c,N*sizeof(int),hipMemcpyDeviceToHost,stream));
    HANDLE_ERROR(hipMemcpyAsync(c+i+N,dev_c1,N*sizeof(int),hipMemcpyDeviceToHost,stream1));
  }
  HANDLE_ERROR(hipEventRecord(stop,0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elaspedtime,start,stop));
  printf("second total time:%.3f ms\n",elaspedtime);
  HANDLE_ERROR(hipStreamSynchronize(stream));
  HANDLE_ERROR(hipStreamSynchronize(stream1));
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));
  HANDLE_ERROR(hipFree(dev_a1));
  HANDLE_ERROR(hipFree(dev_b1));
  HANDLE_ERROR(hipFree(dev_c1));
  HANDLE_ERROR(hipHostFree(a));
  HANDLE_ERROR(hipHostFree(b));
  HANDLE_ERROR(hipHostFree(c));
  HANDLE_ERROR(hipStreamDestroy(stream));
  HANDLE_ERROR(hipStreamDestroy(stream1));
  getchar();
  return 0;
}