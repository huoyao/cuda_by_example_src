#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "..\common\book.h"
#include <hip/hip_runtime.h>
#include ""

#define  SIZE (100*1024*1024)

__global__ void hist_kernel(unsigned char *dev_buf,long size_,unsigned int *hist_count)
{
  __shared__ unsigned int temp[256];
  temp[threadIdx.x]=0;
  long i=threadIdx.x+blockIdx.x*blockDim.x;
  int offerset=blockDim.x*gridDim.x;
  __syncthreads();
  while (i<size_)
  {
    atomicAdd(&temp[dev_buf[i]],1);
    i+=offerset;
  }
  __syncthreads();
  atomicAdd(&hist_count[threadIdx.x],temp[threadIdx.x]);
}

int main(void)
{
  unsigned char *buffer=(unsigned char *)big_random_block(SIZE);
  hipEvent_t start,stop;
  HANDLE_ERROR(hipEventCreate(&start,0));
  HANDLE_ERROR(hipEventCreate(&stop,0));
  HANDLE_ERROR(hipEventRecord(start,0));
  unsigned char *dev_buf;
  unsigned int *dev_hist_count;
  HANDLE_ERROR(hipMalloc((void **)&dev_buf,SIZE));
  HANDLE_ERROR(hipMemcpy(dev_buf,buffer,SIZE,hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMalloc((void **)&dev_hist_count,256*sizeof(int)));
  HANDLE_ERROR(hipMemset(dev_hist_count,0,256*sizeof(int)));
  hipDeviceProp_t prop;
  HANDLE_ERROR(hipGetDeviceProperties(&prop,0));
  int blocks=prop.multiProcessorCount;
  //blocks*2 may make the gpu run in best status 
  hist_kernel<<<blocks*2,256>>>(dev_buf,SIZE,dev_hist_count);
  unsigned int h_hist_count[256];
  HANDLE_ERROR(hipMemcpy(h_hist_count,dev_hist_count,256*sizeof(int),hipMemcpyDeviceToHost));
  float elaspedtime;
  HANDLE_ERROR(hipEventRecord(stop,0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elaspedtime,start,stop));
  printf("total time:%.3f ms\n",elaspedtime);

  long histcount=0;
  for (int i=0;i!=256;++i)
  {
    histcount+=h_hist_count[i];
    //printf("%d ",h_hist_count[i]);
  }
  printf("sum of hist is:%ld \n",histcount);
  for (int i=0;i!=SIZE;++i)
  {
    h_hist_count[buffer[i]]--;
  }
  for (int i=0;i!=256;++i)
  {
    if (h_hist_count[i]!=0)
    {
      printf("failed\n");
      break;
    }
  }
  HANDLE_ERROR(hipEventDestroy(start));
  HANDLE_ERROR(hipEventDestroy(stop));
  HANDLE_ERROR(hipFree(dev_buf));
  HANDLE_ERROR(hipFree(dev_hist_count));
  free(buffer);
  getchar();
  return 0;
}