#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include""

#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)
#define getLastCudaError(msg)  __getLastCudaError (msg, __FILE__, __LINE__)
#define N (55*1024)
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
#undef  assert
#define assert(arg)
#endif

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions
// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err)
  {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
    return ;
  }
}
// This will output the proper error string when calling hipGetLastError
inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err)
  {
    fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
      file, line, errorMessage, (int)err, hipGetErrorString( err ) );
    return ;
  }
}
// end of CUDA Helper Functions

__global__ void VecAdd(float *a,float *b,float *c)
{
  long long i=threadIdx.x+blockIdx.x*blockDim.x;
  while(i<N)
  {
    a[i]=b[i]+c[i];
    //a[i]/=2.0;
    //a[i]/=3.0;
    i+=blockDim.x+gridDim.x;
  }
  //printf("blockdim:%d\n",blockDim.x);
  assert(blockDim.x);
}

__global__ void add(int a,int b ,int *c)
{
  *c=a+b;
}

int main(){

  hipSetDevice(0);
  hipDeviceSynchronize();
  hipDeviceSynchronize();
  float A[N],B[N],C[N];
  for(long long i=0;i!=N;++i)
  {
      B[i]=i;
      C[i]=i;
  }
  float *dec_a,*dec_b,*dec_c;
  checkCudaErrors( hipMalloc((void**) &dec_a, sizeof(int)*N));
  checkCudaErrors( hipMalloc((void**) &dec_b, sizeof(int)*N));
  checkCudaErrors( hipMalloc((void**) &dec_c, sizeof(int)*N));
  checkCudaErrors( hipMemcpy(dec_b,B,sizeof(int)*N,hipMemcpyHostToDevice));
  checkCudaErrors( hipMemcpy(dec_c,C,sizeof(int)*N,hipMemcpyHostToDevice));
  VecAdd<<<256,128>>>(dec_a,dec_b,dec_c);
  checkCudaErrors( hipMemcpy(A,dec_a,sizeof(int)*N,hipMemcpyDeviceToHost));
  hipFree(dec_a);
  hipFree(dec_b);
  hipFree(dec_c);
  bool suc=true;
  for (long long i=0;i!=N;++i)
  {
    if (A[i]!=B[i]+C[i])
    {
      suc=false;
    }
  }
  if (suc)
  {
    printf("we did it\n");
  } 
  else
  {
    printf("we fail\n");
  }
  //matAdd<<<1,dimBlock>>>(A,B,C);
  int c;
  int *resultc;
  checkCudaErrors( hipMalloc((void**) &resultc, sizeof(int)));
  add<<<1,1>>>(2,7,resultc);
  checkCudaErrors( hipMemcpy(&c,resultc,sizeof(int),hipMemcpyDeviceToHost));
  printf("%d\n",c);
  hipFree(resultc);
  getchar();
  hipDeviceReset();
  return 0;
}