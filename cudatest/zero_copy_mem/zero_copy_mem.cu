#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "..\common\book.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#define  N 200*1024
#define imin(a,b) (a<b?a:b)

const int threadPerBlock=256;
const int blockPerGrid=imin(32,(N+threadPerBlock-1)/threadPerBlock);

__global__ void dot(float *a,float *b,float *c)
{
  __shared__ float cahe[threadPerBlock];
  int tid=threadIdx.x+blockDim.x*blockIdx.x;
  int caheidex=threadIdx.x;
  float temp=0;
  while(tid<N)
  {
    temp+=a[tid]*b[tid];
    tid+=blockDim.x*gridDim.x;
  }
  cahe[caheidex]=temp;
  __syncthreads();
  int i=blockDim.x/2;
  while(i!=0)
  {
    if (caheidex<i)
    {
      cahe[caheidex]+=cahe[caheidex+i];
    }
    __syncthreads();
    i/=2;
  }
  if (caheidex==0)
  {
    c[blockIdx.x]=cahe[0];
  }
}

void host_malloc()
{
  float *a,*b,*c;
  double sum=0.;
  float *dev_a,*dev_b,*dev_c;
  hipEvent_t start,stop;
  float elaspedtime;
  HANDLE_ERROR(hipEventCreate(&start,0));
  HANDLE_ERROR(hipEventCreate(&stop,0));
  a=(float *)malloc(N*sizeof(float));
  b=(float *)malloc(N*sizeof(float));
  c=(float *)malloc(blockPerGrid*sizeof(float));
  HANDLE_ERROR(hipMalloc((void **)&dev_a,N*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&dev_b,N*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&dev_c,blockPerGrid*sizeof(float)));
  for (int i=0;i!=N;++i)
  {
    a[i]=(float)i;
    b[i]=(float)i*2;
  }
  HANDLE_ERROR(hipEventRecord(start,0));
  HANDLE_ERROR(hipMemcpy(dev_a,a,N*sizeof(float),hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b,b,N*sizeof(float),hipMemcpyHostToDevice));
  dot<<<blockPerGrid,threadPerBlock>>>(dev_a,dev_b,dev_c);
  HANDLE_ERROR(hipMemcpy(c,dev_c,blockPerGrid*sizeof(float),hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipEventRecord(stop,0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elaspedtime,start,stop));
  printf("host malloc total time:%.3f ms\n",elaspedtime);
  for (int i=0;i!=blockPerGrid;++i)
  {
    sum+=c[i];
  }
#define sumaryofx(x) (x*(x+1)*(2*x+1)/6)
  printf("does %.3lf=%.3lf\n",sum,2*sumaryofx((double)(N-1)));
  double x=2*1024.;
  double  xx=2*x*(x+1)*(2*x+1)/6;
  printf("%.3lf\n",xx);
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));
  free(a);
  free(b);
  free(c);
}

void zero_copy()
{
  float *a,*b,*c;
  double sum=0.;
  float *dev_a,*dev_b,*dev_c;
  hipEvent_t start,stop;
  float elaspedtime;
  HANDLE_ERROR(hipEventCreate(&start,0));
  HANDLE_ERROR(hipEventCreate(&stop,0));
  HANDLE_ERROR(hipHostAlloc(&a,N*sizeof(float),hipHostMallocMapped|hipHostMallocWriteCombined));
  HANDLE_ERROR(hipHostAlloc(&b,N*sizeof(float),hipHostMallocMapped|hipHostMallocWriteCombined));
  HANDLE_ERROR(hipHostAlloc(&c,blockPerGrid*sizeof(float),hipHostMallocMapped|hipHostMallocWriteCombined));
  for (int i=0;i!=N;++i)
  {
    a[i]=(float)i;
    b[i]=(float)i*2;
  }
  HANDLE_ERROR(hipHostGetDevicePointer(&dev_a,a,0));
  HANDLE_ERROR(hipHostGetDevicePointer(&dev_b,b,0));
  HANDLE_ERROR(hipHostGetDevicePointer(&dev_c,c,0));
  HANDLE_ERROR(hipEventRecord(start,0));
  dot<<<blockPerGrid,threadPerBlock>>>(dev_a,dev_b,dev_c);
  HANDLE_ERROR(hipDeviceSynchronize());
  HANDLE_ERROR(hipEventRecord(stop,0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elaspedtime,start,stop));
  printf("\nzero copy total time:%.3f ms\n",elaspedtime);
  for (int i=0;i!=blockPerGrid;++i)
  {
    sum+=c[i];
  }
#define sumaryofx(x) (x*(x+1)*(2*x+1)/6)
  printf("does %.3lf=%.3lf\n",sum,2*sumaryofx((double)(N-1)));
  double x=2*1024.;
  double  xx=2*x*(x+1)*(2*x+1)/6;
  printf("%.3lf\n",xx);
  HANDLE_ERROR(hipHostFree(a));
  HANDLE_ERROR(hipHostFree(b));
  HANDLE_ERROR(hipHostFree(c));
}

int main(void)
{
  hipDeviceProp_t prop;
  int dev;
  HANDLE_ERROR(hipGetDevice(&dev));
  HANDLE_ERROR(hipGetDeviceProperties(&prop,dev));
  if (prop.canMapHostMemory!=1)
  {
    printf("device can`t map memory\n");
    getchar();
    return 0;
  }
  host_malloc();
  zero_copy();
  HANDLE_ERROR(hipDeviceReset());
  if (getchar()==27)
  {
    exit(0);
  }
  return 0;
}