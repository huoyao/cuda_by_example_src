#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "..\common\book.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#define  N 2*1024
#define imin(a,b) (a<b?a:b)

const int threadPerBlock=256;
const int blockPerGrid=imin(32,(N+threadPerBlock-1)/threadPerBlock);

struct datastruct
{
  int deviceid;
  int size;
  float *a;
  float *b;
  float returnvalue;
};
__global__ void dot(float *a,float *b,float *c)
{
  __shared__ float cahe[threadPerBlock];
  int tid=threadIdx.x+blockDim.x*blockIdx.x;
  int caheidex=threadIdx.x;
  float temp=0;
  while(tid<N)
  {
    temp+=a[tid]*b[tid];
    tid+=blockDim.x*gridDim.x;
  }
  cahe[caheidex]=temp;
  __syncthreads();
  int i=blockDim.x/2;
  while(i!=0)
  {
    if (caheidex<i)
    {
      cahe[caheidex]+=cahe[caheidex+i];
    }
    __syncthreads();
    i/=2;
  }
  if (caheidex==0)
  {
    c[blockIdx.x]=cahe[0];
  }
}

void *rutine(void *pvoiddata)
{
  datastruct *data=(datastruct *)pvoiddata;
  HANDLE_ERROR(hipSetDevice(data->deviceid));
  int size=data->size;
  float *a,*b,*c;
  double sum=0.;
  float *dev_a,*dev_b,*dev_c;
  hipEvent_t start,stop;
  float elaspedtime;
  HANDLE_ERROR(hipEventCreate(&start,0));
  HANDLE_ERROR(hipEventCreate(&stop,0));
  a=data->a;
  b=data->b;
  c=(float *)malloc(blockPerGrid*sizeof(float));
  HANDLE_ERROR(hipMalloc((void **)&dev_a,size*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&dev_b,size*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void **)&dev_c,blockPerGrid*sizeof(float)));
  HANDLE_ERROR(hipEventRecord(start,0));
  HANDLE_ERROR(hipMemcpy(dev_a,a,size*sizeof(float),hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b,b,size*sizeof(float),hipMemcpyHostToDevice));
  dot<<<blockPerGrid,threadPerBlock>>>(dev_a,dev_b,dev_c);
  HANDLE_ERROR(hipMemcpy(c,dev_c,blockPerGrid*sizeof(float),hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipEventRecord(stop,0));
  HANDLE_ERROR(hipEventSynchronize(stop));
  HANDLE_ERROR(hipEventElapsedTime(&elaspedtime,start,stop));
  printf("host malloc total time:%.3f ms\n",elaspedtime);
  for (int i=0;i!=blockPerGrid;++i)
  {
    sum+=c[i];
  }
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));
  free(a);
  free(b);
  free(c);
  data->returnvalue=sum;
  return 0;
}

int main(void)
{
  int dev_count;
  HANDLE_ERROR(hipGetDeviceCount(&dev_count));
  if (dev_count<2)
  {
    printf("no more gpu,gpu num��%d\n",dev_count);
    getchar();
    return 0;
  }
  float *a=(float *)malloc(N*sizeof(float));
  HANDLE_NULL(a);
  float *b=(float *)malloc(N*sizeof(float));
  HANDLE_NULL(b);
  for (int i=0;i!=N;++i)
  {
    a[i]=(float)i;
    b[i]=(float)i*2;
  }
  datastruct data[2];
  data[0].deviceid=0;
  data[0].a=a;
  data[0].b=b;
  data[0].size=N/2;
  data[1].deviceid=1;
  data[1].a=a+N/2;
  data[1].b=b+N/2;
  data[1].size=N/2;
  CUTThread threadx=start_thread((CUT_THREADROUTINE)rutine,&data[0]);
  rutine(&data[1]);
  end_thread(threadx);
  free(a);
  free(b);
  printf("value:%f\n",data[0].returnvalue+data[1].returnvalue);
  hipDeviceReset();
  if (getchar()==27)
  {
    exit(0);
  }
  return 0;
}